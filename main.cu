#include "hip/hip_runtime.h"

#include "def.cuh"

using namespace H5;

void print(std::ostream& os, const energy_type& x)
{
  os << x;
}

template <typename Array>
void print(std::ostream& os, const Array& A)
{
  typename Array::const_iterator i;
  os << "[";
  for (i = A.begin(); i != A.end(); i++) {
    print(os, *i);
    if (boost::next(i) != A.end())
      os << ',';
  }
  os << "]" << endl;
}

vector<energy_type> span(energy_type minimum, energy_type maximum, unsigned len) {
    energy_type step = (maximum - minimum) / (len - 1), curr = minimum;
    vector<energy_type> ret(len);
    for (unsigned i = 0; i < len; ++i) {
        ret[i] = curr;
        curr += step;
    }
    return ret;
}

/*
   Save multi_array to hdf5 file
 */
int save2Hdf5(Array3D& x, Array3D& y, Array3D& z, Array3D& edepavg)
{
    const H5std_string  FILE_NAME("edep.hdf5");

    try
        {
            /*
             * Turn off the auto-printing when failure occurs so that we can
             * handle the errors appropriately
             */
            H5::Exception::dontPrint();
            /*
             * Create a new file using H5F_ACC_TRUNC access,
             * default file creation properties, and default file
             * access properties.
             */
            H5::H5File file(FILE_NAME, H5F_ACC_TRUNC);
            /*
             * Define the size of the array and create the data space for fixed
             * size dataset.
             */
            hsize_t     dimsf[3];              // dataset dimensions
            dimsf[0] = nx;
            dimsf[1] = ny;
            dimsf[2] = nz;
            H5::DataSpace dataspace(3, dimsf);
            /*
             * Define datatype for the data in the file.
             * We will store little endian INT numbers.
             */
            H5::IntType datatype(H5::PredType::NATIVE_DOUBLE);
            datatype.setOrder(H5T_ORDER_LE);
            /*
             * Create a new dataset within the file using defined dataspace and
             * datatype and default dataset creation properties.
             */
            H5::DataSet dataset = file.createDataSet("/Coordinate_x", datatype, dataspace);
            dataset.write(x.data(), H5::PredType::NATIVE_DOUBLE);
            dataset = file.createDataSet("/Coordinate_y", datatype, dataspace);
            dataset.write(y.data(), H5::PredType::NATIVE_DOUBLE);
            dataset = file.createDataSet("/Coordinate_z", datatype, dataspace);
            dataset.write(z.data(), H5::PredType::NATIVE_DOUBLE);

            dataset = file.createDataSet("/Edepavg", datatype, dataspace);
            /*
             * Write the data to the dataset using default memory space, file
             * space, and transfer properties.
             */
            dataset.write(edepavg.data(), H5::PredType::NATIVE_DOUBLE);
        }  // end of try block
    // catch failure caused by the H5File operations
    catch(H5::Exception& error )
        {
            error.printErrorStack();
            return -1;
        }
    return 0;  // successfully terminated
}

void rayTracing(vector<energy_type> te_profile, vector<energy_type> r_profile, 
        vector<energy_type> ne_profile, energy_type *edep) {

    struct timeval time1, time2, time3, time4, total;
    gettimeofday(&time1, NULL);

    vector<energy_type> phase_r = span(0.0, 0.1, 2001);
    vector<energy_type> pow_r(2001);

#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (unsigned i = 0; i < 2001; ++i) {
        pow_r[i] = exp(-1*pow(pow((phase_r[i]/sigma),2), (5.0/2.0)));
    }
    //printf("Starting CUDA code\n");

    position_type **dev_beam_norm = new position_type *[nGPUs];
    position_type **dev_bbeam_norm = new position_type *[nGPUs];
    energy_type **dev_ne_profile = new energy_type *[nGPUs];
    energy_type **dev_te_profile = new energy_type *[nGPUs];
    energy_type **dev_r_profile = new energy_type *[nGPUs];
    energy_type **dev_pow_r = new energy_type *[nGPUs];
    energy_type **dev_phase_r = new energy_type *[nGPUs];

    position_type *better_beam_norm = new position_type[nbeams*4];
    for (int b = 0; b < nbeams; ++b) {
        position_type theta1 = acos(beam_norm[b][2]);
        position_type theta2 = atan2(beam_norm[b][1]*focal_length, beam_norm[b][0]*focal_length);
        better_beam_norm[4*b] = cos(theta1);
        better_beam_norm[4*b+1] = sin(theta1);
        better_beam_norm[4*b+2] = cos(theta2);
        better_beam_norm[4*b+3] = sin(theta2);
    }

    energy_type **dev_edep = new energy_type *[nGPUs];
    energy_type **edep_per_GPU = new energy_type *[nGPUs];
    for (int i = 0; i < nGPUs; ++i) {
        edep_per_GPU[i] = (energy_type *)malloc(sizeof(energy_type)*edep_size);

        safeGPUAlloc((void **)&dev_beam_norm[i], sizeof(position_type)*3*nbeams, i);
        safeGPUAlloc((void **)&dev_bbeam_norm[i], sizeof(position_type)*4*nbeams, i);
        safeGPUAlloc((void **)&dev_pow_r[i], sizeof(energy_type)*2001, i);
        safeGPUAlloc((void **)&dev_phase_r[i], sizeof(energy_type)*2001, i);
        safeGPUAlloc((void **)&dev_ne_profile[i], sizeof(energy_type)*nr, i);
        safeGPUAlloc((void **)&dev_te_profile[i], sizeof(energy_type)*nr, i);
        safeGPUAlloc((void **)&dev_r_profile[i], sizeof(energy_type)*nr, i);
        safeGPUAlloc((void **)&dev_edep[i], sizeof(energy_type)*edep_size, i);
    
        moveToAndFromGPU(dev_beam_norm[i], &(beam_norm[0][0]), sizeof(position_type)*3*nbeams, i);
        moveToAndFromGPU(dev_bbeam_norm[i], &(better_beam_norm[0]), sizeof(position_type)*4*nbeams, i);
        moveToAndFromGPU(dev_pow_r[i], &(pow_r[0]), sizeof(energy_type)*2001, i);
        moveToAndFromGPU(dev_phase_r[i], &(phase_r[0]), sizeof(energy_type)*2001, i);
        moveToAndFromGPU(dev_ne_profile[i], &(ne_profile[0]), sizeof(energy_type)*nr, i);
        moveToAndFromGPU(dev_te_profile[i], &(te_profile[0]), sizeof(energy_type)*nr, i);
        moveToAndFromGPU(dev_r_profile[i], &(r_profile[0]), sizeof(energy_type)*nr, i);
    }

    gettimeofday(&time2, NULL);

    position_type grad_const = pow(c, 2) / (2.0 * ncrit) * dt * 0.5;
    position_type dedx_const = grad_const / xres;
    position_type dedy_const = grad_const / yres;
    position_type dedz_const = grad_const / zres;

    dim3 nblocks(nbeams/nGPUs, threads_per_beam/threads_per_block, 1);
    //printf("%d %d\n", nbeams/nGPUs, threads_per_beam);
    //printf("%d\n", nindices);

    // We put the launches in their own loops for timing purposes
#ifdef USE_OPENMP
#pragma omp parallel for num_threads (nGPUs)
#endif
    for (int i = 0; i < nGPUs; ++i) { 
        hipSetDevice(i);
        launch_ray_XYZ<<<nblocks, threads_per_block>>>(i, nindices, dev_te_profile[i], dev_r_profile[i],
            dev_ne_profile[i], dev_edep[i], dev_bbeam_norm[i],
            dev_beam_norm[i], dev_pow_r[i], dev_phase_r[i],
            dedx_const, dedy_const, dedz_const);
        hipDeviceSynchronize();
    }
    
    for (int i = 0; i < nGPUs; ++i) {
        moveToAndFromGPU(edep_per_GPU[i], dev_edep[i], sizeof(energy_type)*edep_size, i);
        hipFree(dev_pow_r[i]);
        hipFree(dev_phase_r[i]);
        hipFree(dev_beam_norm[i]);
        hipFree(dev_bbeam_norm[i]);
        hipFree(dev_te_profile[i]);
        hipFree(dev_ne_profile[i]);
        hipFree(dev_r_profile[i]);
        hipFree(dev_edep[i]);
    }
    delete[] dev_pow_r;
    delete[] dev_phase_r;
    delete[] dev_beam_norm;
    delete[] dev_bbeam_norm;
    delete[] dev_te_profile;
    delete[] dev_ne_profile;
    delete[] dev_r_profile;
    delete[] dev_edep;

    gettimeofday(&time3, NULL);
    for (int l = 0; l < nGPUs; ++l) {
#ifdef USE_OPENMP
#pragma omp parallel for
#endif
        for (long i = 0; i < nx+2; ++i) {
            for (long j = 0; j < ny+2; ++j) {
                for (long k = 0; k < nz+2; ++k) {
                    edep[i*(ny+2)*(nz+2) + j*(nz+2) + k] += edep_per_GPU[l][i*(ny+2)*(nz+2) + j*(nz+2) + k];
                }
            }
        }
    }
    for (int i = 0; i < nGPUs; ++i) {
        free(edep_per_GPU[i]);
    }

    delete[] edep_per_GPU;

    hipDeviceReset();

    gettimeofday(&time4, NULL);
    timersub(&time4, &time1, &total);
    timersub(&time4, &time3, &time4);
    timersub(&time3, &time2, &time3);
    timersub(&time2, &time1, &time2);
#ifndef PRINT
    printf("rt: Init %ld.%06ld\nTracing %ld.%06ld\nCombining %ld.%06ld\n"
           "Total %ld.%06ld\n",
           time2.tv_sec, time2.tv_usec,
           time3.tv_sec, time3.tv_usec,
           time4.tv_sec, time4.tv_usec,
           total.tv_sec, total.tv_usec);
#endif
}

int main(int argc, char **argv) {

#ifdef USE_OPENMP
    if (argc <= 1) {
        omp_set_num_threads(1);
    } else {
        omp_set_num_threads(atoi(argv[1]));
    }
#endif

    // r_data is the radius profile, ne_data is the electron profile
    // and te_data is the temperature profile
    vector<energy_type> r_data(nr), te_data(nr), ne_data(nr);

    // Load data from files
    fstream file;
    file.open("./s83177_wCBET_t301_1p5ns_te.txt");
    for (unsigned i = 0; i < nr; ++i)
        file >> r_data[i] >> te_data[i];
    file.flush();
    file.close();
    file.open("./s83177_wCBET_t301_1p5ns_ne.txt");
    for (unsigned i = 0; i < nr; ++i) {
        file >> r_data[i] >> ne_data[i];
    }
    file.flush();
    file.close();

    Array3D edep(boost::extents[nx+2][ny+2][nz+2]);

    rayTracing(te_data, r_data, ne_data, &edep[0][0][0]);
/*
    Array3D edepavg(boost::extents[nx][ny][nz]);
    Array3D x(boost::extents[nx][ny][nz]);
    Array3D y(boost::extents[nx][ny][nz]);
    Array3D z(boost::extents[nx][ny][nz]);
#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            for (int k = 0; k < nz; ++k) {
                x[i][j][k] = i*dx+xmin;
                y[i][j][k] = j*dy+ymin;
                z[i][j][k] = k*dz+zmin;
            }
        }
    }

#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (unsigned i = 0; i < nx; ++i) {
        for (unsigned j = 0; j < ny; ++j) {
            for (unsigned k = 0; k < nz; ++k) {
                edepavg[i][j][k] =
                    (edep[i][j][k] + edep[i+1][j][k] + edep[i+2][j][k] +
                     edep[i][j+1][k] + edep[i+1][j+1][k] + edep[i+2][j+1][k] +
                     edep[i][j+2][k] + edep[i+1][j+2][k] + edep[i+2][j+2][k] +
                     edep[i][j][k+1] + edep[i+1][j][k+1] + edep[i+2][j][k+1] +
                     edep[i][j+1][k+1] + edep[i+1][j+1][k+1] + edep[i+2][j+1][k+1] +
                     edep[i][j+2][k+1] + edep[i+1][j+2][k+1] + edep[i+2][j+2][k+1] +
                     edep[i][j][k+2] + edep[i+1][j][k+2] + edep[i+2][j][k+2] +
                     edep[i][j+1][k+2] + edep[i+1][j+1][k+2] + edep[i+2][j+1][k+2] +
                     edep[i][j+2][k+2] + edep[i+1][j+2][k+2] + edep[i+2][j+2][k+2]) / 27;
            }
        }
    }

    save2Hdf5(x, y, z, edepavg);
*/
#ifdef PRINT
    print(std::cout, edep);
#endif
    return 0;
}
