#include "hip/hip_runtime.h"

#include "def.cuh"

using namespace H5;

double interp(const vector<double> y, const vector<double> x, const double xp)
{
    unsigned low, high, mid;
    assert(x.size() == y.size());

    if (x.front() <= x.back()) {
        // x monotonically increase
        if (xp <= x.front())
            return y.front();
        else if (xp >= x.back())
            return y.back();

        low = 0;
        high = x.size() - 1;
        mid = (low + high) >> 1;
        while (low < high - 1) { if (x[mid] >= xp) high = mid;
            else
                low = mid;
            mid = (low + high) >> 1;
        }

        assert((xp >= x[mid]) && (xp <= x[mid + 1]));
        return y[mid] +
            (y[mid + 1] - y[mid]) / (x[mid + 1] - x[mid]) * (xp - x[mid]);
    } else {
        // x monotonically decrease
        if (xp >= x.front())
            return y.front();
        else if (xp <= x.back())
            return y.back();

        low = 0;
        high = x.size() - 1;
        mid = (low + high) >> 1;
        while (low < high - 1) {
            if (x[mid] <= xp)
                low = mid;
            else
                high = mid;
            mid = (low + high) >> 1;
        }

        assert((xp <= x[mid]) && (xp >= x[mid + 1]));
        return y[mid] +
            (y[mid +1] - y[mid]) / (x[mid + 1] - x[mid]) * (xp - x[mid]);
    }
}

void print(std::ostream& os, const double& x)
{
  os << x;
}

template <typename Array>
void print(std::ostream& os, const Array& A)
{
  typename Array::const_iterator i;
  os << "[";
  for (i = A.begin(); i != A.end(); i++) {
    print(os, *i);
    if (boost::next(i) != A.end())
      os << ',';
  }
  os << "]" << endl;
}

vector<double> span(double minimum, double maximum, unsigned len) {
    double step = (maximum - minimum) / (len - 1), curr = minimum;
    vector<double> ret(len);
    for (unsigned i = 0; i < len; ++i) {
        ret[i] = curr;
        curr += step;
    }
    return ret;
}

/*
   Save multi_array to hdf5 file
 */
int save2Hdf5(Array3D& x, Array3D& y, Array3D& z, Array3D& edepavg)
{
    const H5std_string  FILE_NAME("edep.hdf5");

    try
        {
            /*
             * Turn off the auto-printing when failure occurs so that we can
             * handle the errors appropriately
             */
            H5::Exception::dontPrint();
            /*
             * Create a new file using H5F_ACC_TRUNC access,
             * default file creation properties, and default file
             * access properties.
             */
            H5::H5File file(FILE_NAME, H5F_ACC_TRUNC);
            /*
             * Define the size of the array and create the data space for fixed
             * size dataset.
             */
            hsize_t     dimsf[3];              // dataset dimensions
            dimsf[0] = nx;
            dimsf[1] = ny;
            dimsf[2] = nz;
            H5::DataSpace dataspace(3, dimsf);
            /*
             * Define datatype for the data in the file.
             * We will store little endian INT numbers.
             */
            H5::IntType datatype(H5::PredType::NATIVE_DOUBLE);
            datatype.setOrder(H5T_ORDER_LE);
            /*
             * Create a new dataset within the file using defined dataspace and
             * datatype and default dataset creation properties.
             */
            H5::DataSet dataset = file.createDataSet("/Coordinate_x", datatype, dataspace);
            dataset.write(x.data(), H5::PredType::NATIVE_DOUBLE);
            dataset = file.createDataSet("/Coordinate_y", datatype, dataspace);
            dataset.write(y.data(), H5::PredType::NATIVE_DOUBLE);
            dataset = file.createDataSet("/Coordinate_z", datatype, dataspace);
            dataset.write(z.data(), H5::PredType::NATIVE_DOUBLE);

            dataset = file.createDataSet("/Edepavg", datatype, dataspace);
            /*
             * Write the data to the dataset using default memory space, file
             * space, and transfer properties.
             */
            dataset.write(edepavg.data(), H5::PredType::NATIVE_DOUBLE);
        }  // end of try block
    // catch failure caused by the H5File operations
    catch(H5::Exception& error )
        {
            error.printErrorStack();
            return -1;
        }
    return 0;  // successfully terminated
}

void rayTracing(Array3D eden, Array3D etemp, 
        double *ne_profile, double *edep, int *marked, int *boxes, double *coverage, int *counter) {

    struct timeval time1, time2, time3, time4, total;
    gettimeofday(&time1, NULL);

    vector<double> phase_r = span(0.0, 0.1, 2001);
    vector<double> pow_r(2001);

#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (unsigned i = 0; i < 2001; ++i) {
        pow_r[i] = exp(-1*pow(pow((phase_r[i]/sigma),2), (5.0/2.0)));
    }
    //printf("Starting CUDA code\n");

    double **dev_beam_norm = new double *[nGPUs];
    double **dev_bbeam_norm = new double *[nGPUs];
    double **dev_ne_profile = new double *[nGPUs];
    double **dev_te_profile = new double *[nGPUs];
    double **dev_r_profile = new double *[nGPUs];
    double **dev_pow_r = new double *[nGPUs];
    double **dev_phase_r = new double *[nGPUs];

	double **dev_eden = new double *[nGPUs];
 	double **dev_etemp = new double *[nGPUs];

    int **dev_marked = new int *[nGPUs];
    int **dev_boxes = new int *[nGPUs];

  	double **dev_coverage = new double *[nGPUs];

    double *better_beam_norm = new double[nbeams*4];
    for (int b = 0; b < nbeams; ++b) {
        double theta1 = acos(beam_norm[b][2]);
        double theta2 = atan2(beam_norm[b][1]*focal_length, beam_norm[b][0]*focal_length);
        better_beam_norm[4*b] = cos(theta1);
        better_beam_norm[4*b+1] = sin(theta1);
        better_beam_norm[4*b+2] = cos(theta2);
        better_beam_norm[4*b+3] = sin(theta2);
    }

    double **dev_edep = new double *[nGPUs];
    double **edep_per_GPU = new double *[nGPUs];
    for (int i = 0; i < nGPUs; ++i) {
        edep_per_GPU[i] = (double *)malloc(sizeof(double)*edep_size);

        safeGPUAlloc((void **)&dev_beam_norm[i], sizeof(double)*3*nbeams, i);
        safeGPUAlloc((void **)&dev_bbeam_norm[i], sizeof(double)*4*nbeams, i);
        safeGPUAlloc((void **)&dev_pow_r[i], sizeof(double)*2001, i);
        safeGPUAlloc((void **)&dev_phase_r[i], sizeof(double)*2001, i);
        safeGPUAlloc((void **)&dev_ne_profile[i], sizeof(double)*nr, i);
        safeGPUAlloc((void **)&dev_te_profile[i], sizeof(double)*nr, i);
        safeGPUAlloc((void **)&dev_r_profile[i], sizeof(double)*nr, i);
        safeGPUAlloc((void **)&dev_edep[i], sizeof(double)*edep_size, i);
        //safeGPUAlloc((void **)&dev_marked[i], sizeof(int)*nx*ny*nz*2020, i);
        safeGPUAlloc((void **)&dev_boxes[i], sizeof(int)*nbeams*nrays*ncrossings*3, i);
        safeGPUAlloc((void **)&dev_coverage[i], sizeof(double)*nbeams*nrays*ncrossings, i);
		    safeGPUAlloc((void **)&dev_eden[i], sizeof(double)*nx*ny*nz, i);
		    safeGPUAlloc((void **)&dev_etemp[i], sizeof(double)*nx*ny*nz, i);
    
        moveToAndFromGPU(dev_beam_norm[i], &(beam_norm[0][0]), sizeof(double)*3*nbeams, i);
        moveToAndFromGPU(dev_bbeam_norm[i], &(better_beam_norm[0]), sizeof(double)*4*nbeams, i);
        moveToAndFromGPU(dev_pow_r[i], &(pow_r[0]), sizeof(double)*2001, i);
        moveToAndFromGPU(dev_phase_r[i], &(phase_r[0]), sizeof(double)*2001, i);
        /*moveToAndFromGPU(dev_ne_profile[i], &(ne_profile[0]), sizeof(double)*nr, i);
        moveToAndFromGPU(dev_te_profile[i], &(te_profile[0]), sizeof(double)*nr, i);
        moveToAndFromGPU(dev_r_profile[i], &(r_profile[0]), sizeof(double)*nr, i);*/
		    moveToAndFromGPU(dev_eden[i], &(eden[0][0][0]), sizeof(double)*nx*ny*nz, i);
		    moveToAndFromGPU(dev_etemp[i], &(etemp[0][0][0]), sizeof(double)*nx*ny*nz, i);
    }

    gettimeofday(&time2, NULL);

    double grad_const = pow(c, 2) / (2.0 * ncrit) * dt * 0.5;
    double dedx_const = grad_const / xres;
    double dedy_const = grad_const / yres;
    double dedz_const = grad_const / zres;

    dim3 nblocks(nbeams/nGPUs, threads_per_beam/threads_per_block, 1);
    //printf("%d %d\n", nbeams/nGPUs, threads_per_beam);
    //printf("%d\n", nindices);

    // We put the launches in their own loops for timing purposes
#ifdef USE_OPENMP
#pragma omp parallel for num_threads (nGPUs)
#endif
    for (int i = 0; i < nGPUs; ++i) { 
        hipSetDevice(1);
        launch_ray_XYZ<<<nblocks, threads_per_block>>>(i, nindices, dev_eden[i], dev_etemp[i],
            dev_edep[i], dev_bbeam_norm[i],
            dev_beam_norm[i], dev_pow_r[i], dev_phase_r[i],
            dedx_const, dedy_const, dedz_const, marked, dev_boxes[i], dev_coverage[i], counter);
        cout << hipDeviceSynchronize() << endl;
    }
    
    for (int i = 0; i < nGPUs; ++i) {
        moveToAndFromGPU(edep_per_GPU[i], dev_edep[i], sizeof(double)*edep_size, i);
        moveToAndFromGPU(boxes, dev_boxes[i], sizeof(int)*nbeams*nrays*ncrossings*3, i);
        moveToAndFromGPU(coverage, dev_coverage[i], sizeof(double)*nbeams*nrays*ncrossings, i);
        hipFree(dev_pow_r[i]);
        hipFree(dev_phase_r[i]);
        hipFree(dev_beam_norm[i]);
        hipFree(dev_bbeam_norm[i]);
        hipFree(dev_te_profile[i]);
        hipFree(dev_ne_profile[i]);
        hipFree(dev_r_profile[i]);
        hipFree(dev_edep[i]);
        hipFree(dev_eden[i]);
        hipFree(dev_etemp[i]);
        hipFree(dev_coverage[i]);
        hipFree(dev_boxes[i]);
    }
    delete[] dev_pow_r;
    delete[] dev_phase_r;
    delete[] dev_beam_norm;
    delete[] dev_bbeam_norm;
    delete[] dev_te_profile;
    delete[] dev_ne_profile;
    delete[] dev_r_profile;
    delete[] dev_edep;
    delete[] dev_eden;
    delete[] dev_etemp;
    delete[] dev_coverage;
    delete[] dev_boxes;

    gettimeofday(&time3, NULL);
    for (int l = 0; l < nGPUs; ++l) {
#ifdef USE_OPENMP
#pragma omp parallel for
#endif
        for (long i = 0; i < nx+2; ++i) {
            for (long j = 0; j < ny+2; ++j) {
                for (long k = 0; k < nz+2; ++k) {
                    edep[i*(ny+2)*(nz+2) + j*(nz+2) + k] += edep_per_GPU[l][i*(ny+2)*(nz+2) + j*(nz+2) + k];
                }
            }
        }
    }
    for (int i = 0; i < nGPUs; ++i) {
        free(edep_per_GPU[i]);
    }

    delete[] edep_per_GPU;

    hipDeviceReset();

    gettimeofday(&time4, NULL);
    timersub(&time4, &time1, &total);
    timersub(&time4, &time3, &time4);
    timersub(&time3, &time2, &time3);
    timersub(&time2, &time1, &time2);
#ifndef PRINT
    printf("rt: Init %ld.%06ld\nTracing %ld.%06ld\nCombining %ld.%06ld\n"
           "Total %ld.%06ld\n",
           time2.tv_sec, time2.tv_usec,
           time3.tv_sec, time3.tv_usec,
           time4.tv_sec, time4.tv_usec,
           total.tv_sec, total.tv_usec);
#endif
}

int main(int argc, char **argv) {

    if (argc <= 1) {
    } else {
        omp_set_num_threads(atoi(argv[1]));
    }

    // r_data is the radius profile, ne_data is the electron profile
    // and te_data is the temperature profile
    vector<double> r_data(nr), te_data(nr), ne_data(nr);

    // Load data from files
    fstream file;
    file.open("./s83177_wCBET_t301_1p5ns_te.txt");
    for (unsigned i = 0; i < nr; ++i)
        file >> r_data[i] >> te_data[i];
    file.flush();
    file.close();
    file.open("./s83177_wCBET_t301_1p5ns_ne.txt");
    for (unsigned i = 0; i < nr; ++i) {
        file >> r_data[i] >> ne_data[i];
    }
    file.flush();
    file.close();

    Array3D edep(boost::extents[nx+2][ny+2][nz+2]),
		  etemp(boost::extents[nx][nz][nz]),
      eden(boost::extents[nx][nz][nz]);

    int *marked, *counter;

    // 1200 here is temporary, need to come up with bound
    hipMallocManaged(&marked, sizeof(int)*nx*ny*nz*1200);
    hipMallocManaged(&counter, sizeof(int)*nx*ny*nz);

    Array4I boxes(boost::extents[nbeams][nrays][ncrossings][3]);
    Array3D area_coverage(boost::extents[nbeams][nrays][ncrossings]);
    
#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (unsigned i = 0; i < nx; ++i) {
        for (unsigned j = 0; j < ny; ++j) {
            for (unsigned k = 0; k < nz; ++k) {
                double tmp = sqrt(pow(i*dx+xmin, 2) + pow(j*dy+ymin, 2) + pow(k*dz+zmin, 2));
                eden[i][j][k] = interp(ne_data, r_data, tmp);
                //etemp[i][j][k] = 5.2e-5 * 10.0 / pow(interp(te_data, r_data, tmp), 1.5);
                etemp[i][j][k] = interp(te_data, r_data, tmp);
                //wpe[i][j][k] = sqrt((pow(omega, 2) - pow(sqrt(eden[i][j][k]*1e6*pow(ec,2)/((double)me*e0)), 2)) /
                 //       pow(c, 2.0));
                //wpe[i][j][k] = sqrt(eden[i][j][k]*1e6*pow(ec,2)/((double)me*e0));
            }
        }
    }
    rayTracing(eden, etemp, NULL, &edep[0][0][0], marked, &boxes[0][0][0][0], &area_coverage[0][0][0], counter);
    // Below is the code for writing to the hdf5 file
    /*
    Array3D edepavg(boost::extents[nx][ny][nz]);
    Array3D x(boost::extents[nx][ny][nz]);
    Array3D y(boost::extents[nx][ny][nz]);
    Array3D z(boost::extents[nx][ny][nz]);
#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            for (int k = 0; k < nz; ++k) {
                x[i][j][k] = i*dx+xmin;
                y[i][j][k] = j*dy+ymin;
                z[i][j][k] = k*dz+zmin;
            }
        }
    }

#ifdef USE_OPENMP
#pragma omp parallel for
#endif
    for (unsigned i = 0; i < nx; ++i) {
        for (unsigned j = 0; j < ny; ++j) {
            for (unsigned k = 0; k < nz; ++k) {
                edepavg[i][j][k] =
                    (edep[i][j][k] + edep[i+1][j][k] + edep[i+2][j][k] +
                     edep[i][j+1][k] + edep[i+1][j+1][k] + edep[i+2][j+1][k] +
                     edep[i][j+2][k] + edep[i+1][j+2][k] + edep[i+2][j+2][k] +
                     edep[i][j][k+1] + edep[i+1][j][k+1] + edep[i+2][j][k+1] +
                     edep[i][j+1][k+1] + edep[i+1][j+1][k+1] + edep[i+2][j+1][k+1] +
                     edep[i][j+2][k+1] + edep[i+1][j+2][k+1] + edep[i+2][j+2][k+1] +
                     edep[i][j][k+2] + edep[i+1][j][k+2] + edep[i+2][j][k+2] +
                     edep[i][j+1][k+2] + edep[i+1][j+1][k+2] + edep[i+2][j+1][k+2] +
                     edep[i][j+2][k+2] + edep[i+1][j+2][k+2] + edep[i+2][j+2][k+2]) / 27;
            }
        }
    }

    save2Hdf5(x, y, z, edepavg);
*/
    // For quick testing
#ifdef PRINT
    print(std::cout, edep);
#endif
    return 0;
}
